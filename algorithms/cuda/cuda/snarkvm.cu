#include "hip/hip_runtime.h"
// Copyright 2024 Aleo Network Foundation
// This file is part of the snarkVM library.

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at:

// http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381.hpp>
#elif defined(FEATURE_BLS12_377)
# include <ff/bls12-377.hpp>
#elif defined(FEATURE_BN254)
# include <ff/alt_bn128.hpp>
#else
# error "no FEATURE"
#endif

#include <ntt/ntt.cuh>
#include "polynomial.cuh"

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_inf_t affine_t;
typedef fr_t scalar_t;

#define WBITS 11 // Suitable for ~2^16
#include <msm/pippenger.cuh>
#undef WBITS

// A simple way to allocate a host pointer without having to
// care about freeing it.
template<typename T> class host_ptr_t {
    T* h_ptr;
public:
    host_ptr_t(size_t nelems) : h_ptr(nullptr)
    {
        if (nelems) {
            CUDA_OK(hipHostMalloc(&h_ptr, nelems * sizeof(T)));
        }
    }
    ~host_ptr_t() { if (h_ptr) hipHostFree((void*)h_ptr); }

    inline operator const T*() const            { return h_ptr; }
    inline operator T*() const                  { return h_ptr; }
    inline operator void*() const               { return (void*)h_ptr; }
    inline const T& operator[](size_t i) const  { return h_ptr[i]; }
    inline T& operator[](size_t i)              { return h_ptr[i]; }
};

#ifndef __CUDA_ARCH__

#include <vector>
#include <chrono>
#include <atomic>

typedef std::chrono::high_resolution_clock Clock;

using namespace std;

class snarkvm_t {
    thread_pool_t pool;

    struct resource_t {
        int dev;
        int stream;
        resource_t(int _dev, int _stream) {
            dev = _dev;
            stream = _stream;
        }
    };
    channel_t<resource_t*> resources;
    
    // Host and device storage
    size_t                         allocated_elements;
    std::vector<dev_ptr_t<fr_t>*>  d_mem;
    std::vector<host_ptr_t<fr_t>*> h_mem;

    void free_memory() {
        allocated_elements = 0;
        for (size_t i = 0; i < ngpus(); i++) {
            if (d_mem[i] != nullptr) {
                delete d_mem[i];
                d_mem[i] = nullptr;
            }
            if (h_mem[i] != nullptr) {
                delete h_mem[i];
                h_mem[i] = nullptr;
            }
        }
    }

    fr_t *d_addr(uint32_t dev, uint32_t stream) {
        if (allocated_elements == 0) {
            assert(false);
            return nullptr;
        }
        fr_t* dmem = *d_mem[dev];
        return &dmem[allocated_elements * stream];
    }
    fr_t *h_addr(uint32_t dev, uint32_t stream) {
        if (allocated_elements == 0) {
            assert(false);
            return nullptr;
        }
        fr_t* hmem = *h_mem[dev];
        return &hmem[allocated_elements * stream];
    }
    
public:
    snarkvm_t(int max_lg_domain) : pool(ngpus()) {
        // Allocate storage for 4 polynomials, required by polynomial multiplication
        // Will be allocated per gpu per stream
        allocated_elements = ((size_t)1 << max_lg_domain) * 4;

        d_mem.resize(ngpus());
        h_mem.resize(ngpus());
        for (size_t i = 0; i < ngpus(); i++) {
            d_mem[i] = nullptr;
            h_mem[i] = nullptr;
        }
        try {
            for (size_t i = 0; i < ngpus(); i++) {
                auto& gpu = select_gpu(i);
                d_mem[i] = new dev_ptr_t<fr_t>(allocated_elements * gpu_t::FLIP_FLOP);
                h_mem[i] = new host_ptr_t<fr_t>(allocated_elements * gpu_t::FLIP_FLOP);
            }
        } catch (const cuda_error& e) {
            // Failed to allocate. Clean up any memory
            free_memory();
        }            

        // GPU resource allocation scheme
        for (size_t j = 0; j < gpu_t::FLIP_FLOP; j++) {
            for (size_t dev = 0; dev < ngpus(); dev++) {
                resources.send(new resource_t(dev, j));
            }
        }
    }
    ~snarkvm_t() {}

    RustError NTT(fr_t* out, fr_t* in,
                  uint32_t lg_domain_size,
                  NTT::InputOutputOrder ntt_order,
                  NTT::Direction ntt_direction,
                  NTT::Type ntt_type) {
        size_t domain_size = (size_t)1 << lg_domain_size;

        // Ensure enough sufficient memory
        if (allocated_elements < domain_size) {
            return RustError{hipErrorOutOfMemory};
        }

        resource_t* resource = resources.recv();
        int dev = resource->dev;
        auto& gpu = select_gpu(dev);
        int stream_idx = resource->stream;
        stream_t& stream = gpu[stream_idx];

        fr_t* h_mem = h_addr(dev, stream_idx);
        memcpy(h_mem, in, sizeof(fr_t) * domain_size);

        // Perform NTT
        RustError e = Polynomial::Base(gpu, h_mem, lg_domain_size,
                                       ntt_order, ntt_direction, ntt_type);
        if (e.code != hipSuccess) {
            resources.send(resource);
            return e;
        }
        // Successful. Copy the result back
        memcpy(out, h_mem, sizeof(fr_t) * domain_size);
        resources.send(resource);
        return RustError{hipSuccess};
    }

    RustError PolyMul(fr_t* out,
                      size_t pcount, fr_t** polynomials, size_t* plens,
                      size_t ecount, fr_t** evaluations, size_t* elens,
                      uint32_t lg_domain_size) {
        // domain_size is the size of the final polynomial
        size_t domain_size = (size_t)1 << lg_domain_size;

        // Corner cases
        if (pcount + ecount == 0) {
            return RustError{hipSuccess};
        } else if (pcount + ecount == 1) {
            if (pcount == 1) {
                memcpy(out, polynomials[0], sizeof(fr_t) * plens[0]);
            } else {
                // Perform iNTT on the single evaluation
                memset((uint8_t*)out, 0, sizeof(fr_t) * domain_size);
                memcpy(out, evaluations[0], sizeof(fr_t) * elens[0]);
                return NTT(out, out, lg_domain_size,
                           NTT::InputOutputOrder::NN, NTT::Direction::inverse,
                           NTT::Type::standard);
            }
            return RustError{hipSuccess};
        }
        
        // Ensure enough sufficient memory
        if (allocated_elements < 4 * domain_size) {
            return RustError{hipErrorOutOfMemory};
        }

        resource_t* resource = resources.recv();
        int dev = resource->dev;
        auto& gpu = select_gpu(dev);
        int stream_idx = resource->stream;
        stream_t& stream = gpu[stream_idx];

        // // Copy data to pinned staging buffer
        fr_t* h_mem0 = h_addr(dev, stream_idx);
        fr_t* h_mem1 = &h_mem0[domain_size];
        fr_t* h_mem2 = &h_mem0[domain_size * 2];
        fr_t* d_mem0 = d_addr(dev, stream_idx);
        fr_t* d_mem1 = &d_mem0[domain_size];
        fr_t* d_mem2 = &d_mem0[domain_size * 2];
        fr_t* d_mem3 = &d_mem0[domain_size * 3];
        RustError e = Polynomial::Mul(gpu, stream,
                                      h_mem0, h_mem1, h_mem2,
                                      d_mem0, d_mem1, d_mem2, d_mem3,
                                      pcount, polynomials, plens,
                                      ecount, evaluations, elens,
                                      lg_domain_size);
        if (e.code != hipSuccess) {
            resources.send(resource);
            return e;
        }

        hipDeviceSynchronize();
        memcpy(out, h_mem0, sizeof(fr_t) * domain_size);

        resources.send(resource);
        return RustError{hipSuccess};
    }

    RustError MSM(point_t* out, const affine_t points[], size_t npoints,
                  const scalar_t scalars[], size_t ffi_affine_size) {
        // SNP TODO: cleanup
        // auto start = Clock::now();

        size_t gpu_count = min(ngpus(), npoints);
        point_t partial_sums[gpu_count];
        size_t bases_per_gpu = (npoints + gpu_count - 1) / gpu_count;
        channel_t<size_t> ch;
        RustError error = RustError{hipSuccess};

        // Divide the MSM among the GPUs
        for (size_t i = 0; i < gpu_count; i++) {
            pool.spawn([&, i]() {
                int dev = i;
                select_gpu(dev);
                size_t start = i * bases_per_gpu;
                size_t sz = std::min(bases_per_gpu, npoints - start);

                // This is ugly, but we only know the size of the affine points in bytes
                const affine_t* pts = (affine_t*)(&((uint8_t*)points)[start * ffi_affine_size]);
                
                RustError ret;
                try {
                    msm_t<bucket_t, point_t, affine_t, scalar_t> msm(dev);
                    ret = msm.invoke(partial_sums[i], slice_t<affine_t>{pts, sz},
                                     &scalars[start], false, ffi_affine_size);
                } catch (const cuda_error& e) {
                    out->inf();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
                    ret = RustError{e.code(), e.what()};
#else
                    ret = RustError{e.code()};
#endif
                }
                if (ret.code != hipSuccess) {
                    error = ret;
                }
                ch.send(i);
            });
        }
        size_t dev = ch.recv();
        *out = partial_sums[dev];
        for (size_t i = 0; i < gpu_count - 1; i++) {
            dev = ch.recv();
            point_t::dadd(*out, *out, partial_sums[dev]);
        }
        // auto end = Clock::now();
        // uint64_t dt = std::chrono::duration_cast<
        //     std::chrono::microseconds>(end - start).count();
        // printf("MSM size %ld took %ld us\n", npoints, dt);

        return error;
        
        // auto start = Clock::now();
        // auto res = mult_pippenger<bucket_t>(out, points, npoints, scalars,
        //                                     false, ffi_affine_size);
        // auto end = Clock::now();
        // uint64_t dt = std::chrono::duration_cast<
        //     std::chrono::microseconds>(end - start).count();
        // printf("MSM size %ld took %ld us\n", npoints, dt);
        // return res;
    }
};

#endif
