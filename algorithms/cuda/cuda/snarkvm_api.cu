#include "hip/hip_runtime.h"
// Copyright 2024 Aleo Network Foundation
// This file is part of the snarkVM library.

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at:

// http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>

#include "snarkvm.cu"

#ifndef __CUDA_ARCH__

// Lazy instantiation of snarkvm_t
class snarkvm_singleton_t {
    bool failed = false;
    snarkvm_t *snarkvm = nullptr;

public:
    snarkvm_singleton_t() {}
    ~snarkvm_singleton_t() {
        delete snarkvm;
        snarkvm = nullptr;
    }
    bool ok() {
        if (!failed && snarkvm == nullptr) {
            // SNP TODO: max domain size?
            snarkvm = new snarkvm_t(17);
            if (snarkvm == nullptr) {
                failed = true;
            }
        }
        return snarkvm != nullptr;
    }
    snarkvm_t* operator->() {
        assert (ok());
        return snarkvm;
    }
};
snarkvm_singleton_t snarkvm_g;
                                         
#ifndef __CUDA_ARCH__

extern "C" {
    RustError snarkvm_ntt(fr_t* inout, uint32_t lg_domain_size,
                          NTT::InputOutputOrder ntt_order, NTT::Direction ntt_direction,
                          NTT::Type ntt_type)
    {
        if (!snarkvm_g.ok()) {
            return RustError{hipErrorOutOfMemory};
        }
        return snarkvm_g->NTT(inout, inout, lg_domain_size, ntt_order,
                              ntt_direction, ntt_type);
    }

    RustError snarkvm_polymul(fr_t* out,
                              size_t pcount, fr_t** polynomials, size_t* plens,
                              size_t ecount, fr_t** evaluations, size_t* elens,
                              uint32_t lg_domain_size) {
        if (!snarkvm_g.ok()) {
            return RustError{hipErrorOutOfMemory};
        }
        return snarkvm_g->PolyMul(out,
                                  pcount, polynomials, plens,
                                  ecount, evaluations, elens,
                                  lg_domain_size);
    }

    RustError snarkvm_msm(point_t* out, const affine_t points[], size_t npoints,
                          const scalar_t scalars[], size_t ffi_affine_size) {
        if (!snarkvm_g.ok()) {
            return RustError{hipErrorOutOfMemory};
        }
        return snarkvm_g->MSM(out, points, npoints, scalars, ffi_affine_size);
    }
}
#endif // __CUDA_ARCH__

#endif
